#include "hip/hip_runtime.h"
// $ nvcc -std=c++11 -I../.. graph_daxpy.cu -o graph_daxpy
#include <cassert>
#include <iostream>
#include <chrono>
#include <thrust/device_vector.h>

__global__ void daxpy_kernel(int n, double a, const double* x, double* y)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if(i < n)
  {
    y[i] = a * x[i] + y[i];
  }
}

__global__ void hello_world(int n, double a)
{
  if(blockIdx.x == 0 and threadIdx.x == 0) 
  {
    printf("Hello world!\n");
    printf("n: %d\n", n);
    printf("a: %f\n", a);
  }
}

hipGraph_t make_daxpy_graph(int n, double a, const double* x, double* y)
{
  hipGraph_t graph{};
  if(hipError_t error = hipGraphCreate(&graph, 0))
  {
    throw std::runtime_error("make_daxpy_graph: CUDA error after hipGraphCreate: " + std::string(hipGetErrorString(error)));
  }

  int block_size = 256;
  int num_blocks = (n + block_size - 1) / block_size;

  // introduce a kernel node
  void* kernel_params[] = {&n, &a, &x, &y};

  hipKernelNodeParams params
  {
    reinterpret_cast<void*>(daxpy_kernel),
    dim3(num_blocks),
    dim3(block_size),
    0,
    kernel_params,
    nullptr
  };

  hipGraphNode_t node{};
  if(hipError_t error = hipGraphAddKernelNode(&node, graph, nullptr, 0, &params))
  {
    throw std::runtime_error("make_daxpy_graph: CUDA error after hipGraphAddKernelNode: " + std::string(hipGetErrorString(error)));
  }

  return graph;
}

void test(size_t n)
{
  thrust::device_vector<double> x(n, 1);
  thrust::device_vector<double> y(n, 2);
  double a = 2;

  // make the graph
  hipGraph_t graph = make_daxpy_graph(n, a, x.data().get(), y.data().get());

  // instantiate the graph
  hipGraphExec_t executable_graph{};
  if(hipError_t error = hipGraphInstantiate(&executable_graph, graph, nullptr, nullptr, 0))
  {
    throw std::runtime_error("test: CUDA error after hipGraphInstantiate: " + std::string(hipGetErrorString(error)));
  }

  // create a stream
  hipStream_t stream{};
  if(hipError_t error = hipStreamCreate(&stream))
  {
    throw std::runtime_error("test: CUDA error after hipStreamCreate: " + std::string(hipGetErrorString(error)));
  }

  // launch the graph
  if(hipError_t error = hipGraphLaunch(executable_graph, stream))
  {
    throw std::runtime_error("test: CUDA error after hipGraphLaunch: " + std::string(hipGetErrorString(error)));
  }

  // wait
  if(hipError_t error = hipStreamSynchronize(stream))
  {
    throw std::runtime_error("test: CUDA error after hipStreamSynchronize: " + std::string(hipGetErrorString(error)));
  }

  // destroy resources
  if(hipError_t error = hipStreamDestroy(stream))
  {
    throw std::runtime_error("test: CUDA error after hipStreamDestroy: " + std::string(hipGetErrorString(error)));
  }
  
  if(hipError_t error = hipGraphDestroy(graph))
  {
    throw std::runtime_error("test: CUDA error after hipGraphDestroy: " + std::string(hipGetErrorString(error)));
  }

  if(hipError_t error = hipGraphExecDestroy(executable_graph))
  {
    throw std::runtime_error("test: CUDA error after hipGraphExecDestroy: " + std::string(hipGetErrorString(error)));
  }

  // check the result
  thrust::device_vector<double> reference(n, 4);
  assert(reference == y);
}

double measure_bandwidth(size_t n, size_t num_trials = 100)
{
  thrust::device_vector<double> x(n, 1);
  thrust::device_vector<double> y(n, 2);
  double a = 2;

  // make a stream
  hipStream_t stream;
  if(hipError_t error = hipStreamCreate(&stream))
  {
    throw std::runtime_error("measure_bandwidth: CUDA error after hipStreamCreate: " + std::string(hipGetErrorString(error)));
  }

  // make the graph
  hipGraph_t graph = make_daxpy_graph(n, a, x.data().get(), y.data().get());

  // instantiate the graph
  hipGraphExec_t executable_graph;
  if(hipError_t error = hipGraphInstantiate(&executable_graph, graph, nullptr, nullptr, 0))
  {
    throw std::runtime_error("test: CUDA error after hipGraphInstantiate: " + std::string(hipGetErrorString(error)));
  }

  // time trials
  auto start = std::chrono::high_resolution_clock().now();
  {
    for(size_t i = 0; i < num_trials; ++i)
    {
      hipGraphLaunch(executable_graph, stream);
    }

    if(hipError_t error = hipStreamSynchronize(stream))
    {
      throw std::runtime_error("measure_bandwidth: CUDA error after hipStreamSynchronize: " + std::string(hipGetErrorString(error)));
    }
  }
  auto end = std::chrono::high_resolution_clock().now();

  // compute mean GB/s
  size_t mean_nanoseconds = (std::chrono::duration_cast<std::chrono::nanoseconds>(end - start) / num_trials).count();
  double mean_seconds = double(mean_nanoseconds) / 1000000000;
  size_t num_bytes = 2 * n * sizeof(double);
  double mean_bytes_per_second = double(num_bytes) / mean_seconds;
  double mean_gigabytes_per_second = mean_bytes_per_second / 1000000000;

  if(hipError_t error = hipGraphExecDestroy(executable_graph))
  {
    throw std::runtime_error("measure_bandwidth: CUDA error after hipGraphExecDestroy: " + std::string(hipGetErrorString(error)));
  }

  if(hipError_t error = hipGraphDestroy(graph))
  {
    throw std::runtime_error("measure_bandwidth: CUDA error after hipGraphDestroy: " + std::string(hipGetErrorString(error)));
  }

  if(hipError_t error = hipStreamDestroy(stream))
  {
    throw std::runtime_error("measure_bandwidth: CUDA error after hipStreamDestroy: " + std::string(hipGetErrorString(error)));
  }

  return mean_gigabytes_per_second;
}

int main(int argc, char** argv)
{
  size_t n = 1 << 25;
  if(argc > 1)
  {
    n = std::atoi(argv[1]);
  }

  // first test for correctness
  test(n);

  double bandwidth = measure_bandwidth(n);

  std::clog << n << ", " << bandwidth << std::endl;
  std::cout << "Graph DAXPY bandwidth: " << bandwidth << " GB/s" << std::endl;
  std::cout << "OK" << std::endl;

  return 0;
}


