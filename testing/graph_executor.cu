#include <iostream>
#include <cassert>
#include <graph_executor.hpp>
#include <void_sender.hpp>

int main()
{
  {
    // test A then B

    void_sender start;

    graph_executor ex;

    auto task_a = ex.then_execute(
      [] __host__ __device__ ()
      {
        printf("Task A\n");
      },
      start
    );

    auto task_b = ex.then_execute(
      [] __host__ __device__ ()
      {
        printf("Task B\n");
      },
      task_a
    );

    hipStream_t stream{};
    if(auto error = hipStreamCreate(&stream))
    {
      throw std::runtime_error("CUDA error after hipStreamCreate: " + std::string(hipGetErrorString(error)));
    }

    task_b.submit(stream);

    if(auto error = hipStreamSynchronize(stream))
    {
      throw std::runtime_error("CUDA error after hipStreamSynchronize: " + std::string(hipGetErrorString(error)));
    }

    if(auto error = hipStreamDestroy(stream))
    {
      throw std::runtime_error("CUDA error after hipStreamDestroy: " + std::string(hipGetErrorString(error)));
    }
  }

  std::cout << "OK" << std::endl;
  
  return 0;
}

