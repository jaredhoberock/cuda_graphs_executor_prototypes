#include "hip/hip_runtime.h"
#include <iostream>
#include <cassert>
#include <bulk_graph_executor.hpp>
#include <void_sender.hpp>

__managed__ unsigned int result;

int main()
{
  hipStream_t stream{};
  if(auto error = hipStreamCreate(&stream))
  {
    throw std::runtime_error("CUDA error after hipStreamCreate(): " + std::string(hipGetErrorString(error)));
  }

  {
    // test with
    // * empty outer shared object
    // * empty inner shared object

    bulk_graph_executor ex(stream);

    void_sender start;

    grid_index shape_a{dim3(10), dim3(10)};

    result = 0;

    auto task_a = ex.bulk_then_execute(
      [] __device__ (grid_index idx)
      {
        dim3 block_idx = idx[0];
        dim3 thread_idx = idx[1];

        unsigned int my_contribution = block_idx.x ^ thread_idx.x;

        atomicXor(&result, my_contribution);
      },
      shape_a,
      start
    );

    grid_index shape_b{dim3(10), dim3(10)};

    auto task_b = ex.bulk_then_execute(
      [] __device__ (grid_index idx)
      {
        dim3 block_idx = idx[0];
        dim3 thread_idx = idx[1];

        unsigned int my_contribution = block_idx.x ^ thread_idx.x;

        atomicXor(&result, my_contribution);
      },
      shape_b,
      task_a
    );

    task_b.submit();

    task_b.sync_wait();

    // compute the expected result
    unsigned int expected_result = 0;
    for(unsigned int i = 0; i < shape_a[0].x; ++i)
    {
      for(unsigned int j = 0; j < shape_a[1].x; ++j)
      {
        expected_result ^= (i ^ j);
      }
    }

    for(unsigned int i = 0; i < shape_b[0].x; ++i)
    {
      for(unsigned int j = 0; j < shape_b[1].x; ++j)
      {
        expected_result ^= (i ^ j);
      }
    }

    assert(expected_result == result);
  }

  if(auto error = hipStreamDestroy(stream))
  {
    throw std::runtime_error("CUDA error after hipStreamDestroy(): " + std::string(hipGetErrorString(error)));
  }

  std::cout << "OK" << std::endl;
  
  return 0;
}

